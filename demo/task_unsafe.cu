#include "hip/hip_runtime.h"
#include <iostream>
#include "TheEmployeesSalary.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void gpu_salary_incrementer(const double* original_salary, double* new_salary, int size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size) {
        new_salary[i] = original_salary[i] * 1.15 + 5000;
    }
}

void cpu_salary_incrementer(const double original_salary[], double new_salary[], int size) {
    for (int i = 0; i < size; i++) {
        new_salary[i] = original_salary[i] * 1.15 + 5000;
    }
}

int main() {
    int size = sizeof(TheArrayOfSalaries) / sizeof(double);
    std::cout << "Size of TheArrayOfSalaries : " << size << std::endl;

    // CPU Computation for Reference
    double* cpu_TheArrayOfNewSalaries = new double[size](); // Define an array to hold new salaries, all 0's
    cpu_salary_incrementer(TheArrayOfSalaries, cpu_TheArrayOfNewSalaries, size);

    // GPU Computation
    // 1. Allocation device memory
    double* d_original_salary;
    double* d_new_salary;
    hipMalloc((void**)&d_original_salary, size * sizeof(double));
    hipMalloc((void**)&d_new_salary, size * sizeof(double));

    // 2. Copy data from host to device
    hipMemcpy(d_original_salary, TheArrayOfSalaries, size * sizeof(double), hipMemcpyHostToDevice);

    // 3. Kernel launch
    int threads_per_block = 256;
    int blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;
    gpu_salary_incrementer<<<blocks_per_grid, threads_per_block>>>(d_original_salary, d_new_salary, size);
    hipDeviceSynchronize();

    // 4. Copy data from device to host
    double gpu_TheArrayOfNewSalaries[size] = {0};
    hipMemcpy(gpu_TheArrayOfNewSalaries, d_new_salary, size * sizeof(double), hipMemcpyDeviceToHost);

    // 5. Free device memory)
    hipFree(d_original_salary);
    hipFree(d_new_salary);

    // Compare
    for (int i = 0; i < size; i++) {
        std::cout << TheArrayOfSalaries[i] << " -> " << cpu_TheArrayOfNewSalaries[i] << " = " << gpu_TheArrayOfNewSalaries[i] << std::endl;
    }
    
    // Free host memory
    delete[] cpu_TheArrayOfNewSalaries;
    return 0;
}